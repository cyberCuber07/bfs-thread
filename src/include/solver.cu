#include "hip/hip_runtime.h"
#ifndef __SOLVER_H_
#define __SOLVER_H_

#include <vector>
#include "utils.cu"
#include "../kernels/third.cu"

using namespace DataStructs;


struct BFS {

    int N, max_val, n_workers;
    std::vector<std::pair<int,int>> adj_idxs;
    vec1d<bool> vis;
    vec2d<Edge> adj;

    BFS (std::string path)
    {
        ADJ Adj ( path );
        max_val = Adj.solve();
        // adj = edge2adj<Edge>(reader.load(path), insert, N);
        // sort_adj();
        // vis.assign(N, false);
    }

    /* cuda variables */
    struct ADJ {
        /* GPU struct to:
                -- load memory on global / shared memory
                -- stores data for each node --- adjacency list form
                -- copies data to CPU
         */
        int N, M, * d_M;
        std::size_t size;
        int * d_idxs, * h_idxs;
        bool * d_vis, * h_vis;
        Edge * d_edges, * h_edges;

        void getIdxs(Edge * edges) {
            /* idxs vector stores info when each new node begins in edge list */
            int cnt = 0;
            h_idxs = new int[M];
            for (int i = 1; i < N; ++i) {
                /* compares with "cnt" value because edge numering := {0, ..., max_val - 1} */
                if ( cnt != edges[i].src ) {
                    h_idxs[cnt++] = i;
                    if ( cnt + 1 == M ) return;
                }
            }
        }

        ADJ (std::string path) {
            ReadCSV reader;
            h_edges = reader.load(path, N, M);
            size = N * sizeof(Edge);
            getIdxs(h_edges);
            print( h_idxs, h_idxs + M );
            h_vis = new bool[M];
            for (int i = 0; i < M; ++i) h_vis[i] = false;
            hipMalloc( &d_idxs, M * sizeof(int) );
            hipMalloc( &d_vis, M * sizeof(bool) );
            hipMalloc( &d_edges, size );
            hipMemcpy( d_idxs, h_idxs, M * sizeof(int), hipMemcpyHostToDevice );
            hipMemcpy( d_vis, h_vis, M * sizeof(bool), hipMemcpyHostToDevice );
            hipMemcpy( d_edges, h_edges, size, hipMemcpyHostToDevice );
            // delete[] h_vis;
            // delete[] h_idxs;
            // delete tmp_M;
            // delete[] tmp_edges;
        }

        ~ADJ() {
            delete[] h_vis;
            delete[] h_idxs;
        }

        // ---- BFS ---- //

        int solve() {

            int log_val = M;
            int * d_log, * h_log;
            h_log = new int[log_val];
            for (int i = 0; i < log_val; ++i) h_log[i] = 0;
            hipMalloc( &d_log, sizeof(int) * log_val );

            // main part
            int SIZE = 1024;
            int blockSize = SIZE,
                gridSize = M / SIZE + 1;
            solve_one <<< gridSize, blockSize >>> (d_edges, d_idxs, d_vis, M, N, SIZE, d_log);
            hipDeviceSynchronize();

            // GPU -> CPU
            hipMemcpy( h_log, d_log, sizeof(int) * log_val, hipMemcpyDeviceToHost );
            print( h_log, h_log + M );
            hipFree(d_edges);
            hipFree(d_log);
            hipFree(d_idxs);
            hipFree(d_vis);
            // --------------------------------------------------

            int max_val(0);
            for (int i = 0; i < log_val; ++i) max_val = std::max(max_val, h_log[i]);

            return max_val;
        }
    };

};

#endif
